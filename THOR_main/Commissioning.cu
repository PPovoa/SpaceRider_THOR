#include "THOR.hpp"

void Commissioning_Start(StoredErrors storedErrors, int* previousMode, int* nextMode){

    printf("\nCommissioning mode ====\n");
    
    printf("-Check Connection to SR_MMU\n");
    printf("  Time sync with SR_MMU\n");
    //storedErrors.addError(ErrorHandler::SR_MMU_CantConnect);

    printf("-Check access to OBC_MMU\n");
    //storedErrors.addError(ErrorHandler::OBC_MMU_CantConnect);

    printf("-Check Connection to PDU\n");
    //storedErrors.addError(ErrorHandler::PDU_CantConnect);

    printf("-Monitor PDU\n");
    printf("  Get Output state\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetOuputState);
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetTemperature);
    printf("  Get Current drawn\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetCurrent);
    printf("  Get Voltages levels\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetVoltage);

    printf("-Check connection to the detector unit\n");
    if(Detector_setup()){
        storedErrors.addError(ErrorHandler::DET_CantConnect);
        return;
    }
    
    printf("-Check individual subsystem of detector unit\n");
    //storedErrors.addError(ErrorHandler::DET_Subsytem_Cantconnect);

    printf("-Monitor Detector\n");
    printf("  Get Dead pixels\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetDeadPixels);
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetTemperature);
    printf("  Get Current drawn\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetCurrent);
    printf("  Get Voltages levels\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetVoltage);

    printf("Commissioning done!\n\n");

    *previousMode = *nextMode;
    *nextMode = MODE_HK;
}

int Detector_setup(void){
    int deviceIndex=0;

    // Initializes the Pixet and all connected devices
    int rc = pxcInitialize();
    if (rc) {
        printf("Could not initialize Pixet:\n");
        printErrors("pxcInitialize", rc);
        return 1;
    }

    int connectedDevicesCount = pxcGetDevicesCount();
    printf("Connected devices: %d\n", connectedDevicesCount);

    if (connectedDevicesCount == 0){pxcExit(); return 1;}

    for (unsigned devIdx = 0; (signed)devIdx < connectedDevicesCount; devIdx++){
        char deviceName[256];
        for (int n=0; n<256; n++) deviceName[n]=0;
        pxcGetDeviceName(devIdx, deviceName, 256);

        char chipID[256];
        for (int n=0; n<256; n++) chipID[n]=0;
        pxcGetDeviceChipID(devIdx, 0, chipID, 256);
        printf("Device %d: Name %s, (first ChipID: %s)\n", devIdx, deviceName, chipID);
    }

    pxcLoadDeviceConfiguration(deviceIndex, "./configs/MiniPIX-G05-W0085.xml");

    return 0;
}
#include "THOR.hpp"

void Debug_Start(StoredErrors storedErrors, int* previousMode, int* nextMode){

    printf("\nDebug mode ====\n");

    printf("-Monitor PDU\n");
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetTemperature);
    printf("  Get Current drawn\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetCurrent);
    printf("  Get Voltages levels\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetVoltage);

    printf("-Monitor OBC\n");
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetTemperature);

    // DEFINIR CONDICAO PARA "KNOWN ERROR" !!!!!!!!!!!
    if(isErrorKnown(storedErrors)){ // known error
        ManageErrors(storedErrors, previousMode, nextMode);
        storedErrors.resetErrors();
        /*if(isAffecting_SCI_operations(storedErrors)){
            if(isErrorBearable(storedErrors)){
                printf("Emergency message to Ground\n");
                *previousMode = *nextMode;
                *nextMode = MODE_OBS;
            }
            else{
                *previousMode = *nextMode;
                *nextMode = MODE_HK;
            }
        }
        else{
            *previousMode = *nextMode;
            *nextMode = MODE_OBS;
        }*/
        
    }
    else{ // unkown error
        if(isAffecting_PL_operations(storedErrors)){
            printf("Emergency message to Ground\n");
            *previousMode = *nextMode;
            *nextMode = MODE_HK;
            storedErrors.resetErrors();
            return;
        }

        if(isErrorBearable(storedErrors)){
            printf("Emergency message to Ground\n");
            *previousMode = *nextMode;
            *nextMode = MODE_OBS;
            storedErrors.resetErrors();
            return;
        }
        
        printf("Emergency message to Ground\n");
        *previousMode = *nextMode;
        *nextMode = MODE_HK;
        storedErrors.resetErrors();
    }
}

void ManageErrors(StoredErrors storedErrors, int* previousMode, int* nextMode){

    std::vector<std::pair<ErrorHandler::ErrorSource, uint16_t>> CurrErrors = storedErrors.getThrownErrors();

    for (const auto& error : CurrErrors) {
        //ErrorHandler::ErrorSource errorSource = error.first;
        uint16_t errorCode = error.second;

        switch (errorCode)
        {
        case ErrorHandler::SR_MMU_CantConnect:
            printf("SR_MMU error\n");
            break;
        case ErrorHandler::OBC_MMU_CantConnect:
            printf("OBC_MMU error\n");
            break;
        case ErrorHandler::PDU_CantConnect:
            printf("PDU error\n");
            break;
        case ErrorHandler::DET_CantConnect:
            printf("DET error\n");
            break;
        case ErrorHandler::DET_Subsytem_Cantconnect:
            printf("DET_Subsys error\n");
            break;
        case ErrorHandler::PDU_ErrorGetOuputState:
            printf("PDU_GetOuputState error\n");
            break;
        case ErrorHandler::PDU_ErrorGetTemperature:
            printf("PDU_GetTemperature error\n");
            break;
        case ErrorHandler::PDU_ErrorGetCurrent:
            printf("PDU_GetCurrent error\n");
            break;
        case ErrorHandler::PDU_ErrorGetVoltage:
            printf("PDU_GetVoltage error\n");
            break;
        case ErrorHandler::DET_ErrorGetDeadPixels:
            printf("DET_GetDeadPixels error\n");
            break;
        case ErrorHandler::DET_ErrorGetTemperature:
            printf("DET_GetTemperature error\n");
            break;
        case ErrorHandler::DET_ErrorGetCurrent:
            printf("DET_GetCurrent error\n");
            break;
        case ErrorHandler::DET_ErrorGetVoltage:
            printf("DET_GetVoltage error\n");
            break;            

        default:
            printf("????????\n\n");
            break;
        }

        storedErrors.showErrors();
        
        *previousMode = *nextMode;
        *nextMode = EXIT_PROG;//MODE_HK;
    }
}


int isErrorKnown(StoredErrors storedErrors){

    std::vector<std::pair<ErrorHandler::ErrorSource, uint16_t>> CurrErrors = storedErrors.getThrownErrors();

    for (const auto& error : CurrErrors) {
        //ErrorHandler::ErrorSource errorSource = error.first;
        uint16_t errorCode = error.second;
        if (errorCode == 0) return 0;
    }
    return 1;
}

// to finish
int isAffecting_SCI_operations(StoredErrors storedErrors){
    return 0;
}

// to finish
int isAffecting_PL_operations(StoredErrors storedErrors){
    return 0;
}

// to finish
int isErrorBearable(StoredErrors storedErrors){
    return 0;
}

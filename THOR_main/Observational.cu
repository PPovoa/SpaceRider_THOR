#include "THOR.hpp"

void Observational_Start(StoredErrors storedErrors, int* previousMode, int* nextMode){
    
    int measTime = 5; // Measure time in seconds

    printf("\nObservational mode ====\n");

    printf("OBC turns ON Detector Unit\n"); //storedErrors.addError(ErrorHandler::DET_CantConnect);
    printf("PDUs' power outputs\n"); //storedErrors.addError(ErrorHandler::PDU_CantConnect);
    printf("Detector Unit configuration of operation mode\n"); //storedErrors.addError(ErrorHandler::DET_ConfigMode);

    while(*nextMode == MODE_OBS){
        printf("-Monitor Detector\n");
        printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetTemperature);
        printf("  Get Current drawn\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetCurrent);
        printf("  Get Voltages levels\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetVoltage);

        double DET_temp = 0.0; // Temporary!!!
        double DET_volt = 0.0; // Temporary!!!

        if(DET_temp > DET_MAX_TEMP_THRESH){
            while(DET_temp > DET_MIN_TEMP_THRESH){
                printf("Trun OFF component until certain threshold\n");
                printf("Turn ON component\n");
                printf("report to ground\n");
            }
        }

        if(DET_volt > DET_MAX_VOLT_THRESH){
            printf("Trun OFF components\n");
            printf("report to ground\n");
        }

        printf("If the error is persistence go to Debug mode\n"); // what is consider persistence?
        //*nextMode = MODE_DEBUG;
        //break;

        printf("Scientific Data collection\n");
        printf("  Data storage\n");
        printf("  Data processing\n");
        printf("  Communications with MMU\n");

        
        const char *FilePath = {"./output-files/RawData_5s.t3p"};// Automate name of the file!!!!!!!!!!!
        if(ScientificData(measTime, FilePath))
            storedErrors.addError(ErrorHandler::DET_CantConnect);
        
        printf("If bug identified go to Debug mode\n"); // what is consider a bug?
        *nextMode = MODE_DEBUG;
        //break;
    }
}
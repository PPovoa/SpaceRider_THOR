#include "THOR.hpp"

void Housekeeping_Start(StoredErrors storedErrors, int* previousMode, int error_fromMode, int* nextMode){

    printf("\nHousekeeping mode ====\n");

    printf("Turn off PDU DET Outputs\n");//storedErrors.addError(ErrorHandler::SR_MMU_CantConnect);

    printf("-Monitor PDU\n");
    printf("  Get Output state\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetOuputState);
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetTemperature);
    printf("  Get Current drawn\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetCurrent);
    printf("  Get Voltages levels\n");//storedErrors.addError(ErrorHandler::PDU_ErrorGetVoltage);

    printf("-Monitor Detector\n");
    printf("  Get Temperature\n");//storedErrors.addError(ErrorHandler::DET_ErrorGetTemperature);

    switch (*previousMode)
    {
    case MODE_COMMI:
        if(!storedErrors.hasNoErrors()){
            printf("Emergency message to Ground\n");
            if(!IsErrorManageable(storedErrors.getThrownErrors()))
                printf("Wait for Ground Intervention\n"); // THEN RECEIVE THE NEXT MODE
        }
        *previousMode = *nextMode;
        *nextMode = MODE_OBS;
        printf("Enter observational mode\n");
        break;

    case MODE_DEBUG:
        printf("Emergency message to Ground\n");
        if(IsErrorManageable(storedErrors.getThrownErrors())){
            *previousMode = *nextMode;
            *nextMode = MODE_OBS;
            printf("Enter observational mode\n");
            break;
        }
        else
            printf("Wait for Ground Intervention\n"); // THEN RECEIVE THE NEXT MODE
            *previousMode = *nextMode;
            *nextMode = MODE_TEST;
        break;
        

    case MODE_TEST:
        *previousMode = *nextMode;
        *nextMode = MODE_OBS;
        printf("Enter observational mode\n");
        break;

    case MODE_REBOOT:
        *previousMode = *nextMode;
        *nextMode = MODE_REBOOT;
        printf("Rebooting...\n");
        break;

    default:// ?????????
        printf("default - HK ????????\n\n");
        *previousMode = *nextMode;
        *nextMode = MODE_REBOOT;
        break;
    }
}

// Adaptar funcao
int IsErrorManageable(std::vector<std::pair<ErrorHandler::ErrorSource, uint16_t>> error){
    return 1;
}

int IsAnError(int error){
    if(error != NO_error)
        return 1;
    else
        return 0;
}

#include "THOR.hpp"

std::multimap<std::pair<ErrorHandler::ErrorSource, uint16_t>, bool> StoredErrors::thrownErrors;

int main(){
    int previousMode = 0, nextMode = MODE_COMMI;
    int error = NO_error;
    StoredErrors storedErrors;

    printf("Booting...\n\n");

    while(nextMode != EXIT_PROG){
        switch (nextMode)
        {
        case MODE_COMMI:
            Commissioning_Start(storedErrors, &previousMode, &nextMode);
            // What to do if commissioning has an error??
            break;
        
        case MODE_HK:
            Housekeeping_Start(storedErrors, &previousMode, error, &nextMode);
            break;
        
        case MODE_OBS:
            Observational_Start(storedErrors, &previousMode, &nextMode);
            break;
        
        case MODE_DEBUG:
            Debug_Start(storedErrors, &previousMode, &nextMode);
            nextMode = EXIT_PROG;
            break;
        
        case MODE_TEST://======================
            break;
        
        case MODE_REBOOT:
            printf("Rebooting...\n");
            nextMode = EXIT_PROG;
            break;
        
        default:// ?????????
            printf("default - THOR ????????\n\n");
            break;
        }

        if(!storedErrors.hasNoErrors()){
            nextMode = MODE_DEBUG;
        }
    }

    printf("\nExit Program\n");
    return 0;
}
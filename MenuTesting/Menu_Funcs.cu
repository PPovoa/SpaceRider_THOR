#include "hip/hip_runtime.h"
#include "Menu_Funcs.h"

void Menu_Main(int connectedDevicesCount){
    int opcao = RESET_CYCLE;

    while(opcao != 0){
        do{
            printf("\nDevices connected: %d\n", connectedDevicesCount);
            printf("=====================\n");
            printf("        Menu Main\n");
            printf("=====================\n");
            printf("1 - Show all parameters\n");
            printf("2 - Menu Bias\n");
            printf("3 - Menu Threshold\n");
            printf("4 - Measurement\n");
            printf("5 - Check noisy pixels in a file\n");
            printf("0 - Exit\n");

            printf("\nOption: ");
            std::cin >> opcao;

            if(opcao<0 || opcao>4 || std::cin.fail()){
                std::cin.clear(); //clear bad input flag
                std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
                std::cout << "Invalid input; Please try again.\n";
                opcao = RESET_CYCLE;
            }
        }while(opcao == RESET_CYCLE);

        switch(opcao){
            case 1:
                (void)Show_allParameters(ChooseDevice(connectedDevicesCount));
                WaitforUser();
                break;
            case 2:
                (void)Menu_Bias(connectedDevicesCount);
                break;
            case 3:
                (void)Menu_Threshold(connectedDevicesCount);
                break;
            case 4:
                (void)Start_Measurement(ChooseDevice(connectedDevicesCount));
                break;
            case 5:
                (void)checkNoisyPixelsInFile(ChooseDevice(connectedDevicesCount));
                break;
            case 0:
                pxcExit();
                return;
        }
    }
}

void Menu_Bias(int connectedDevicesCount){
    int opcao = RESET_CYCLE;

    while(opcao != 0){
        do{
            printf("\nDevices connected: %d\n", connectedDevicesCount);
            printf("=================\n");
            printf("    Menu Bias\n");
            printf("=================\n");
            printf("1 - Check Bias\n");
            printf("2 - Change Bias (in progress...)\n");
            printf("0 - Exit\n");

            printf("\nOption: ");
            std::cin >> opcao;

            if(opcao<0 || opcao>2 || std::cin.fail()){
                std::cin.clear(); //clear bad input flag
                std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
                std::cout << "Invalid input; Please try again.\n";
                opcao = RESET_CYCLE;
            }

        }while(opcao == RESET_CYCLE);

        switch(opcao){
            case 1:
                (void)Show_Bias(ChooseDevice(connectedDevicesCount));
                break;
            case 2:
                break;
            case 0:
                return;
        }
        WaitforUser();
    }
}

void Menu_Threshold(int connectedDevicesCount){
    int opcao = RESET_CYCLE;

    while(opcao != 0){
        do{
            printf("\nDevices connected: %d\n", connectedDevicesCount);
            printf("=================\n");
            printf(" Menu Threshold\n");
            printf("=================\n");
            printf("1 - Check Threshold\n");
            printf("2 - Change Threshold (in progress...)\n");
            printf("0 - Exit\n");

            printf("\nOption: ");
            std::cin >> opcao;

            if(opcao<0 || opcao>2 || std::cin.fail()){
                std::cin.clear(); //clear bad input flag
                std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
                std::cout << "Invalid input; Please try again.\n";
                opcao = RESET_CYCLE;
            }

        }while(opcao == RESET_CYCLE);

        switch(opcao){
            case 1:
                (void)Show_Threshold(ChooseDevice(connectedDevicesCount));
                break;
            case 2:
                break;
            case 0:
                return;
        }
        WaitforUser();
    }
}


#include "hip/hip_runtime.h"
#include "Funcs.h"

int SetupDetector(){
    int rc = pxcInitialize();
    if (rc) {
        printf("Could not initialize Pixet:\n");
        printErrors("pxcInitialize", rc);
        exit(0);
    }

    int connectedDevicesCount = pxcGetDevicesCount();
    printf("Connected devices: %d\n", connectedDevicesCount);

    if (connectedDevicesCount == 0){pxcExit(); exit(0);}

    for (unsigned devIdx = 0; (signed)devIdx < connectedDevicesCount; devIdx++){
        char deviceName[256];
        for (int n=0; n<256; n++) deviceName[n]=0;
        pxcGetDeviceName(devIdx, deviceName, 256);

        char chipID[256];
        for (int n=0; n<256; n++) chipID[n]=0;
        pxcGetDeviceChipID(devIdx, 0, chipID, 256);
        printf("Device %d: Name %s, (first ChipID: %s)\n", devIdx, deviceName, chipID);
    }
    
    return connectedDevicesCount;
}

void Show_allParameters(unsigned deviceIndex){
    Show_Temp(deviceIndex);
    Show_Bias(deviceIndex);
    Show_Threshold(deviceIndex);
    //Show_DACs(deviceIndex);
}

void Start_Measurement(unsigned deviceIndex){
    int totalTime, intervalTime; //Measure time in seconds
    std::string folderName;

    do{
        printf("Enter total measure time in secs: ");
        std::cin >> totalTime;

        if(totalTime<=0 || std::cin.fail()){
            std::cin.clear(); //clear bad input flag
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
            std::cout << "Invalid input; Please try again.\n";
            totalTime = 0;
        }
    }while(totalTime == 0);


    do{
        printf("Enter duration of each interval in secs: ");
        std::cin >> intervalTime;

        if(intervalTime<=0 || std::cin.fail()){
            std::cin.clear(); //clear bad input flag
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
            std::cout << "Invalid input; Please try again.\n";
            intervalTime = 0;
        }
    }while(intervalTime == 0);


    printf("Enter name of folder: ");
    std::cin >> folderName;

    timepix3DataDriven(deviceIndex, totalTime, intervalTime, folderName);
}


void WaitforUser(){
    std::cin.clear(); //clear bad input flag
    std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
    std::cout << "Press any key to continue...";
    std::cin.get();
}

unsigned ChooseDevice(int connectedDevicesCount){
    unsigned deviceIdx;
    if (connectedDevicesCount==1){
        return 0;
    }
    else{
        do{
            printf("Choose the device index (from 0 to %d): ", connectedDevicesCount);
            std::cin >> deviceIdx;
        
            if(deviceIdx>=connectedDevicesCount || std::cin.fail()){
                std::cin.clear(); //clear bad input flag
                std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
                std::cout << "Invalid input; Please try again.\n";
                deviceIdx = 99;// reset cycle
            }
        }while(deviceIdx == 99);
        return deviceIdx;
    }
}

// Show_allParameters functions ====================
void Show_Temp(unsigned deviceIndex){
    double val;
    int rc;

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CHIP, &val);
    if(rc)printErrors("pxcGetDeviceParameterDouble - PAR_TEMP_CHIP", rc);
    else printf("CHIP temperature: %f\n", val);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CPU, &val);
    if(rc)printErrors("pxcGetDeviceParameterDouble - PAR_TEMP_CPU", rc);
    else printf("CPU temperature: %f\n", val);

    /*rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CHECK_IN_SW, &val);
    if(rc)printErrors("pxcGetDeviceParameterDouble - PAR_TEMP_CHECK_IN_SW", rc);
    else printf("Max temperature SW: %f\n", val);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CHECK_IN_CPU, &val);
    if(rc)printErrors("pxcGetDeviceParameterDouble - PAR_TEMP_CHECK_IN_CPU", rc);
    else printf("Max temperature CPU: %f\n", val);*/
}

void Show_Bias(unsigned deviceIndex){
    double val;
    int rc;

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_BIAS_SENSE_VOLT, &val);
    if(rc) printErrors("pxcGetDeviceParameterDouble", rc);
    else printf("Bias Voltage (V): %f\n", val);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_BIAS_SENSE_CURR, &val);
    if(rc) printErrors("pxcGetDeviceParameterDouble", rc);
    else printf("Bias Current (uA): %f\n", val);
}

void Show_Threshold(unsigned deviceIndex){
    double val;
    int rc;

    rc = pxcGetThreshold(deviceIndex, 0, &val);
    if(rc) printErrors("pxcGetDeviceParameterDouble", rc);
    else printf("Threshold: %f\n", val);
}

// not in use
void Show_DACs(unsigned deviceIndex){
    unsigned short val;
    int rc;
    unsigned idxChip = 0;
    unsigned width, height;

    int opcao;
    if(pxcGetDeviceChipCount(deviceIndex)>1){
        do{
            rc = pxcGetDeviceDimensions(deviceIndex, &width, &height);
            if(rc) printErrors("pxcGetDeviceDimensions", rc);

            
            printf("Select chip index: ");
            std::cin >> idxChip;

            if(idxChip>=width*height || std::cin.fail()){
                std::cin.clear(); //clear bad input flag
                std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
                std::cout << "Invalid input; Choose between 0 - " << width*height << ". Please try again.\n";
                opcao = RESET_CYCLE;
            }
            
        }while(opcao == RESET_CYCLE);
    }

    rc = pxcGetDAC(deviceIndex, idxChip, PXC_TPX3_IBIAS_PREAMP_ON, &val);
    if(rc) printErrors("pxcGetDAC - PXC_TPX3_IBIAS_PREAMP_ON", rc);
    else printf("DAC of chip %d - PXC_TPX3_IBIAS_PREAMP_ON: %hu\n", idxChip, val);
}

void Change_Bias(unsigned deviceIndex, double *biasVoltage_Set){
    int rc;
    double maxBiasVoltage, minBiasVoltage;

    rc = pxcGetBiasRange(deviceIndex, &minBiasVoltage, &maxBiasVoltage);
    if(rc!=0) printErrors("pxcGetBiasRange", rc);

    do{
        printf("Enter bias voltage in volts: ");
        std::cin >> *biasVoltage_Set;

        if(*biasVoltage_Set < minBiasVoltage || *biasVoltage_Set > maxBiasVoltage || std::cin.fail()){
            std::cin.clear(); //clear bad input flag
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
            std::cout << "Invalid input; Please try again.\n";
        }
    }while(*biasVoltage_Set < minBiasVoltage || *biasVoltage_Set > maxBiasVoltage);

    rc = pxcSetBias(deviceIndex, *biasVoltage_Set);
    if(rc!=0) printErrors("pxcSetBias", rc);

}

void checkNoisyPixelsInFile(unsigned deviceIndex){

    int opcao = RESET_CYCLE;
    DIR *dir = opendir(DIR_OUTPUTFILES);
    struct dirent *entry;
    std::vector<std::string> fileNames;
    std::string extension = ".t3pa";
    int i=0;

    printf("All files in the output directory\n");
    
    if (dir == nullptr ) {
        printf("Error opening output directory\n");
        return;
    }
    
    while ((entry = readdir(dir))) {
        std::string fileName = entry->d_name;
        if(fileName.size() >= extension.size() && fileName.substr(fileName.size() - extension.size()) == extension){
            std::cout << i + 1 << ". " << entry->d_name << "\n";
            fileNames.push_back(fileName);
            i++;
        }
    }

    closedir(dir);

    do{
        printf("\nSelect one file from 1 to %d (-1 to exit): ", i);
        std::cin >> opcao;

        if(opcao == -1) return;

        if(opcao<1 || opcao>i || std::cin.fail()){
            std::cin.clear(); //clear bad input flag
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
            std::cout << "Invalid input; Please try again.\n";
            opcao = RESET_CYCLE;
        }
        
    }while(opcao == RESET_CYCLE);

    auto start = std::chrono::high_resolution_clock::now();
    checkNoisyPixels(deviceIndex, std::string(DIR_OUTPUTFILES) + fileNames[opcao-1]);
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the duration and output it
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    return;
}

// ====================


// Start_Measurement functions ====================
void timepix3DataDriven(unsigned deviceIndex, int totalTime, int intervalTime, std::string folderName){
    int rc; // return codes
    int devIdx = deviceIndex; // transmitted over pointer for use in the callback function 
    
    double biasVoltage_Set;
    Change_Bias(deviceIndex, &biasVoltage_Set);

    // working with TOA, TOATOT, TOT_NOTOA, not working with EVENT_ITOT
    // with TOT_NOTOA be carefully for threshold value
    rc = pxcSetTimepix3Mode(deviceIndex, PXC_TPX3_OPM_TOATOT);
    if(rc!=0) printErrors("pxcSetTimepix3Mode", rc);

    rc = pxcSetDeviceParameter(deviceIndex, PAR_DD_BLOCK_SIZE, 6000);
    printf("pxcSetDeviceParameter %d\n", rc);
    rc = pxcSetDeviceParameter(deviceIndex, PAR_DD_BUFF_SIZE, 100);
    //printf(", %d", rc);
    //rc = pxcSetDeviceParameter(deviceIndex, PAR_DCC_LEVEL, 80); 
    if(rc!=0) printErrors(",", rc);

    // First do the sensor refresh: Clean the chip for free charges.
    // In data-driven/callbacks mode, some chips can sometimes stop producing data
    //    in first measurement, if not refreshed before.
    // Alternatively can be used dummy measurement.
    printf("Refreshing sensor...\n");
    rc = pxcDoSensorRefresh(deviceIndex);
    if(rc!=0) printErrors("pxcDoSensorRefresh", rc);



    // get todays date (YYYY-mm-DD_HH-MM) and insert into files name and measured time
    std::time_t t = std::time(nullptr);
    char date[30];
    std::strftime(date, sizeof(date), "%Y-%m-%d", std::localtime(&t));

    std::string folderDir = std::string(DIR_OUTPUTFILES) + folderName;
    if(mkdir(folderDir.c_str(), 0777) != 0){
        printf("Directory %s failed to create\n", folderDir.c_str());
        return;
    }

    std::string Path_CreateFolder = folderDir + "/RawData";
    if(mkdir(Path_CreateFolder.c_str(), 0777) != 0){
        printf("Directory %s failed to create\n", Path_CreateFolder.c_str());
        return;
    }



    std::string rawFilePath = folderDir + "/RawData/" + folderName;
    std::string HousekeepingFilePath = folderDir + "/Housekeeping_log_" + folderName + "_" + std::string(date) + ".txt";
    //std::string DACFilePath = folderDir + "/DAC_log_" + folderName + "_"  + std::string(date) + ".txt";
    printf("\nFile saved in: %s\n", rawFilePath.c_str());

    printf("Total Time: %d seconds\n", totalTime);
    printf("Time interval: %d seconds\n", intervalTime);

    FILE* HousekeepingFile = std::fopen(HousekeepingFilePath.c_str(), "w");
    if (!HousekeepingFile){
        printf("Error opening the file %s\n", HousekeepingFilePath.c_str());
    }
    Housekeeping_HeaderFile(HousekeepingFile, date, folderName, biasVoltage_Set, totalTime, intervalTime);


    /*FILE* DACFile = std::fopen(DACFilePath.c_str(), "w");
    if (!DACFile){
        printf("Error opening the file %s\n", DACFilePath.c_str());
    }*/


    std::string interationFile;
    std::chrono::steady_clock::time_point clock_begin = std::chrono::steady_clock::now();

    int max_sequences = totalTime/intervalTime;
    for(int i=0; i<max_sequences; i++){
        interationFile = rawFilePath + "_" + std::to_string(i) + ".t3pa";
        //rc = pxcMeasureTpx3DataDrivenMode(deviceIndex, intervalTime, interationFile.c_str(), PXC_TRG_NO, onTpx3Data, (intptr_t)&devIdx);
        rc = pxcMeasureTpx3DataDrivenMode(deviceIndex, intervalTime, interationFile.c_str(), PXC_TRG_NO, 0, (intptr_t)&devIdx);
        if(rc!=0) printErrors("pxcMeasureTpx3DataDrivenMode", rc);

        Housekeeping2File(deviceIndex, HousekeepingFile, clock_begin, i+1, max_sequences, biasVoltage_Set);
        //printf("Sequence %i of %d done.\n", i+1, max_sequences);

        checkNoisyPixels(deviceIndex, interationFile);
    }

    std::fclose(HousekeepingFile);
    //std::fclose(DACFile);  
}

void Housekeeping_HeaderFile(FILE* file, char date[], std::string fileName, double biasVoltage_Set, int totalTime, int intervalTime){

    fprintf(file, "logfile created at %s\n", date);
    fprintf(file, "\nfilename: %s\n", fileName.c_str());
    fprintf(file, "total acquisition time: %d seconds\n", totalTime);
    fprintf(file, "duration of each interval: %d seconds\n", intervalTime);
    fprintf(file, "bias input voltage: %.2f V\n", biasVoltage_Set);

    fprintf(file, "\ntimestamp | chip (ºC) | cpu (ºC) | bias sense voltage (V) | bias sense current (µA)\n");
    printf("\nSequence | timestamp | chip (ºC) | cpu (ºC) | bias sense voltage (V) | bias sense current (µA)\n");
}

void Housekeeping2File(unsigned deviceIndex, FILE* Housekeeping_file, std::chrono::steady_clock::time_point clock_begin, int sequence_num, int max_sequences, double biasVoltage_Set){
    
    auto clock_now = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsedSeconds = clock_now - clock_begin;
    double seconds = elapsedSeconds.count();

    double bias_volt, bias_curr;
    double temp_cpu, temp_chip;
    Get_HousekeepingParams(deviceIndex, &bias_volt, &bias_curr, &temp_cpu, &temp_chip);
    Store_HousekeepingLog(Housekeeping_file, seconds, bias_volt, bias_curr, temp_cpu, temp_chip);
    //Store_DACLog(deviceIndex, DAC_file);

    printf("%3d/%-3d    %-9.3f   %-10.2f  %-8.2f   %-22.2f   %-23.2f\n", sequence_num, max_sequences, seconds, temp_chip, temp_cpu, bias_volt, bias_curr);

    /*printf("Temperature chip: %.2f ºC\n", temp_chip);
    printf("Temperature cpu: %.2f ºC\n", temp_cpu);
    printf("Bias set voltage: %.2f V\n", biasVoltage_Set);
    printf("Bias sense voltage: %.2f V\n", bias_volt);
    printf("Bias sense current: %.2f µA\n", bias_curr);*/
}

void Get_HousekeepingParams(unsigned deviceIndex, double *bias_volt, double *bias_curr, double *temp_cpu, double *temp_chip){
    int rc;

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_BIAS_SENSE_VOLT, bias_volt);
    if(rc) printErrors("Get_HousekeepingParams - PAR_BIAS_SENSE_VOLT", rc);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_BIAS_SENSE_CURR, bias_curr);
    if(rc) printErrors("Get_HousekeepingParams - PAR_BIAS_SENSE_CURR", rc);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CHIP, temp_chip);
    if(rc)printErrors("Get_HousekeepingParams - PAR_TEMP_CHIP", rc);

    rc = pxcGetDeviceParameterDouble(deviceIndex, PAR_TEMP_CPU, temp_cpu);
    if(rc) printErrors("Get_HousekeepingParams - PAR_TEMP_CPU", rc);
}

void Store_HousekeepingLog(FILE* file, double seconds, double bias_volt, double bias_curr, double temp_cpu, double temp_chip){
    std::fprintf(file, "%-9.3f   %-10.2f  %-8.2f   %-22.2f   %-23.2f\n", seconds, temp_chip, temp_cpu, bias_volt, bias_curr);
}

// not in use
void Store_DACLog(unsigned deviceIndex, FILE* file){
    unsigned short value; // u16

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_PREAMP_ON, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_PREAMP_ON\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_PREAMP_OFF, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_PREAMP_OFF\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_VPREAMP_NCAS, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_VPREAMP_NCAS\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_IKRUM, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_IKRUM\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_VFBK, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_VFBK\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_VTHRESHOLD, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_VTHRESHOLD\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_DISCS1_ON, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_DISCS1_ON\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_DISC1_OFF, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_DISC1_OFF\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_DISCS2_ON, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_DISCS2_ON\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_DISCS2_OFF, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_DISCS2_OFF\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_PIXELDAC, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_PIXELDAC\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_TPBUFF_IN, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_TPBUFF_IN\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_TPBUFF_OUT, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_TPBUFF_OUT\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_VTP_COARSE, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_VTP_COARSE\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_VTP_FINE, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_VTP_FINE\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_IBIAS_CP_PLL, &value);
    if(fprintf(file, "%hu ", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_IBIAS_CP_PLL\n");
    }

    pxcGetDAC(deviceIndex, 0, PXC_TPX3_PLL_VCNTRL, &value);
    if(fprintf(file, "%hu\n", value)==0){
        printf("Error saving DAC parameter -> PXC_TPX3_PLL_VCNTRL\n");
    }

}


void checkNoisyPixels(unsigned deviceIndex, std::string fileName){

    int getError, rc;

    uint32_t *index, *d_index;
    uint16_t *tot;
    uint64_t *toa, *d_toa;
    uint8_t *ftoa, *overflow;
    int* d_noisyPixels;

    unsigned width, height;
    rc = pxcGetDeviceDimensions(deviceIndex, &width, &height);
    if(rc){
        printErrors("pxcGetDeviceDimensions", rc);
        return;
    }
    else printf("Width: %d\nHeigth: %d\n", width, height);

    int numPixels = width*height;//256*256;     // Number of distinct index values (number of pixels in detetor)
    
    long numElements = Read_NumLinesFile(fileName);
    int blockSize = 512;
    int gridSize = ceil((double)(numElements + blockSize - 1) / (double)blockSize);

    //printf("\nBlock size: %d\nGrid size: %d\n", blockSize, gridSize);

    MallocCUDA(&index, &tot, &toa, &ftoa, &overflow, &d_index, &d_toa, numElements);

    hipMalloc(&d_noisyPixels, numPixels * sizeof(int));
    hipMemset(d_noisyPixels, 0, numPixels * sizeof(int));

    //printf("Reading %s\n", fileName.c_str());

    getError = Read_TxtRawFile(fileName, index, tot, toa, ftoa, overflow);
    if(getError){
        printf("Error! Read_RawFile()\n");
        return;
    }

    hipMemcpy(d_index, index, numElements * sizeof(uint32_t), hipMemcpyHostToDevice);
    // Launch the kernel to identify noisy pixels
    identifyNoisyPixels<<<gridSize, blockSize>>>(d_index, d_toa, numPixels, d_noisyPixels);

    // Copy noisy pixel counters back to the host
    int* h_NoisyPixels = new int[numPixels];
    hipMemcpy(h_NoisyPixels, d_noisyPixels, numPixels * sizeof(int), hipMemcpyDeviceToHost);

    unsigned char maskMatrix[numPixels];
    rc = pxcGetPixelMaskMatrix(deviceIndex, maskMatrix, sizeof(maskMatrix));
    if(rc!=0) printErrors("pxcGetPixelMaskMatrix", rc);

    // noisy pixel information
    for (int i = 0; i < numPixels; ++i){
        if (h_NoisyPixels[i] >= NOISE_THRESHOLD) {
            printf("Pixel %d is masked with count %d\n" , i, h_NoisyPixels[i]);
            maskMatrix[i]=PXC_PIXEL_MASKED;
            rc = pxcSetPixelMaskMatrix(deviceIndex, maskMatrix, sizeof(maskMatrix));
            if(rc!=0) printErrors("pxcSetPixelMaskMatrix", rc);
        }
    }
   
    // Cleanup: Free memory and release resources
    hipHostFree(index);
    hipHostFree(tot);
    hipHostFree(toa);
    hipHostFree(ftoa);
    hipHostFree(overflow);
    hipHostFree(h_NoisyPixels);
    hipFree(d_index);
    hipFree(d_toa);
    hipFree(d_noisyPixels);
}

__global__
void identifyNoisyPixels(uint32_t *matrixIndices, uint64_t *toaValues, int numEvents, int *NoisyPixels) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < numEvents) {
        int matrixIndex = matrixIndices[tid];
        int toa = toaValues[tid];

        // Iterate over previous events and check if the same index pixel was active within MAX_TOA_DIFF
        for (int i = tid - 1; i >= 0; --i){
            if (matrixIndices[i] == matrixIndex && (toa - toaValues[i]) <= MAX_TOA_DIFF) {
                atomicAdd(&NoisyPixels[matrixIndex], 1); // Increment the noisy pixel counter
                break; // No need to check further, as the condition is satisfied
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

long Read_NumLinesFile(std::string fileName){
    long number_of_lines = 0;
    FILE *infile = fopen(fileName.c_str(), "r");
    int ch;

    if (infile == nullptr)
        printf("Read_NumLinesFile - Error in opening file\n");

    while (EOF != (ch=getc(infile)))
        if ('\n' == ch)
            ++number_of_lines;
    //printf("%u\n", number_of_lines-1);

    return number_of_lines-1;
}

int Read_TxtRawFile(std::string fileName, uint32_t *index, uint16_t *tot, uint64_t *toa, uint8_t *ftoa, uint8_t *overflow){
    FILE *fp;

    fp = fopen(fileName.c_str(), "r");

    if (fp == NULL) {
        printf("Error opening file\n");
        return 1;
    }

    // Ignore first line - header
    char line[100];
    fgets(line, 100, fp);

    // Read remaining lines and convert values to numbers
    int i=0, value;
    while (fscanf(fp, "%d %u %lu %hu %hhu %hhu", &value, &index[i], &toa[i], &tot[i], &ftoa[i], &overflow[i]) == 6){
        i++;
        if (i%10000000==0){
            printf(".");
            fflush(stdout); // colocar se não usar \n
        }
    }
    printf("\n");
    
    fclose(fp);

    return 0;
}

void MallocCUDA(uint32_t **index, uint16_t **tot, uint64_t **toa, uint8_t **ftoa, uint8_t **overflow, uint32_t **d_index, uint64_t **d_toa, long numElements){
    // HOST PINNED
    hipHostMalloc((void**)index, numElements*sizeof(uint32_t));
    hipHostMalloc((void**)tot, numElements*sizeof(uint16_t));
    hipHostMalloc((void**)toa, numElements*sizeof(uint64_t));
    hipHostMalloc((void**)ftoa, numElements*sizeof(uint8_t));
    hipHostMalloc((void**)overflow, numElements*sizeof(uint8_t));

    hipMalloc((void**)d_index, numElements*sizeof(uint32_t));
    hipMalloc((void**)d_toa, numElements*sizeof(uint64_t));
}

// ====================================







// ================================

// callback function for data processing, used by pxcMeasureTpx3DataDrivenMode
void onTpx3Data(intptr_t eventData, intptr_t userData) { // ====================================
    int deviceIndex = *((unsigned*)userData);
    unsigned pixelCount = 0;
    static unsigned long long pixelSum = 0;
    static int calls=0;
    int rc; // return code
    rc = pxcGetMeasuredTpx3PixelsCount(deviceIndex, &pixelCount);
    if(rc!=0) printErrors("pxcGetMeasuredTpx3PixelsCount", rc);

    
    static unsigned char maskMatrix[256*256];
    rc = pxcGetPixelMaskMatrix(deviceIndex, maskMatrix, sizeof(maskMatrix));
    if(rc!=0) printErrors("pxcGetPixelMaskMatrix", rc);


    calls++;
    pixelSum += pixelCount;
    if (eventData!=0) { //eventData!=NULL
        printf("(rc= %d, eventData=(pointer: %lu, view method not defined)) PixelCount: %u PixelSum: %llu\n", rc, (unsigned long)eventData, pixelCount, pixelSum);
    } else {
        printf("(rc= %d, eventData=NULL) calls: %d pxCnt: %u pxSum: %llu\n", rc, calls, pixelCount, pixelSum);
    }
    
    static Tpx3Pixel pxData[1000000];
    rc = pxcGetMeasuredTpx3Pixels(deviceIndex, pxData, 1000000);
    if(rc!=0) printErrors("pxcGetMeasuredTpx3Pixels", rc);

    /*if(!running.load(std::memory_order_relaxed)){
        printf("Aborting Measurement...\n");
        pxcAbortMeasurement(deviceIndex);
    }*/
    /*
    // Check for pixels that were called two time in a row.
    std::vector<int>::iterator it, ls;
    static std::vector<int> OldValues(1000000);
    std::vector<int> DupPixels(1000000);

    std::vector<int> NewValues(pixelCount);
    std::transform(pxData, pxData+pixelCount, NewValues.begin(),
                [](const Tpx3Pixel& pixel) { return pixel.index; });

    std::sort(NewValues.begin(), NewValues.end());

    ls = std::set_intersection(OldValues.begin(), OldValues.end(), NewValues.begin(), NewValues.end(), DupPixels.begin());

    OldValues.clear(); // clear all values
    OldValues.insert(OldValues.begin(), NewValues.begin(), NewValues.end()); // copy NewValues to OldValues
    
    OldValues.erase(std::unique(OldValues.begin(), OldValues.end()), OldValues.end()); //remove duplicates
    //DupPixels.erase(std::unique(DupPixels.begin(), DupPixels.end()), DupPixels.end()); //remove duplicates

    // Print all duplicates
    long int numDuplicates=ls-DupPixels.begin();
    if(numDuplicates != 0){
        pxcAbortMeasurement(deviceIndex); // ABORT MEASUREMENT!!!!!
        printf("\tDuplicates: %ld\n\t", numDuplicates);
        for (it=DupPixels.begin(); it!=ls; ++it){
            //maskMatrix[*it]=PXC_PIXEL_MASKED; // does not work in the middle of the measurement!
            //rc = pxcSetPixelMaskMatrix(deviceIndex, maskMatrix, sizeof(maskMatrix));
            //if(rc!=0) printErrors("pxcSetPixelMatrix", rc);
            printf("%d ", *it);
        }
        printf("\n");
    }
    */
}

// use to show fn name, return code, and last error message
void printErrors(const char* fName, int rc) { // ===============================================
    const int ERRMSG_BUFF_SIZE = 512;
    char errorMsg[ERRMSG_BUFF_SIZE];

    pxcGetLastError(errorMsg, ERRMSG_BUFF_SIZE);
    if (errorMsg[0]>0) {
        printf("%s %d err: %s\n", fName, rc, errorMsg);
    } else {
        printf("%s %d err: ---\n", fName, rc);
    }
}